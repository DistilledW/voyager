#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "hip/hip_runtime.h"
#include ""
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvToolsExt.h>
#include <thrust/host_vector.h>
#include <tuple>
#include "types.h"
#include "runtime_switching.h"

#include <thrust/fill.h>
#include <thrust/reduce.h>
#include <stdio.h>
__global__ void markTargetNodes(Node* nodes, int N, int target, int* node_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = 0;
	Node node = nodes[idx];
	if (node.depth > target)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		Node parentnode = nodes[node.parent];
		if (parentnode.depth > target)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}
	node_counts[idx] = count;
}

__global__ void putRenderIndices(Node* nodes, int N, int* node_counts, int* node_offsets, int* render_indices, int* parent_indices = nullptr, int* nodes_for_render_indices = nullptr)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];
	int count = node_counts[idx];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];
	// int start = node.start;
	
	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		if (parent_indices)
			parent_indices[offset + i] = parentgaussian; 
		if (nodes_for_render_indices)
			nodes_for_render_indices[offset + i] = idx;
	}
}

int Switching::expandToTarget(
	int N,
	int target,
	int* nodes,
	int* render_indices
)
{
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);

	int num_blocks = (N + 255) / 256;
	markTargetNodes << <num_blocks, 256 >> > ((Node*)nodes, N, target, render_counts.data().get());

	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);

	putRenderIndices << <num_blocks, 256 >> > ((Node*)nodes, N, render_counts.data().get(), render_offsets.data().get(), render_indices);

	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

__device__ bool inboxCUDA(Box& box, Point viewpoint)
{
	bool inside = true;
	for (int i = 0; i < 3; i++)
	{
		inside &= viewpoint.xyz[i] >= box.minn.xyz[i] && viewpoint.xyz[i] <= box.maxx.xyz[i];
	}
	return inside;
}

__device__ float pointboxdistCUDA(Box& box, Point viewpoint)
{
	Point closest = {
		max(box.minn.xyz[0], min(box.maxx.xyz[0], viewpoint.xyz[0])),
		max(box.minn.xyz[1], min(box.maxx.xyz[1], viewpoint.xyz[1])),
		max(box.minn.xyz[2], min(box.maxx.xyz[2], viewpoint.xyz[2]))
	};

	Point diff = {
		viewpoint.xyz[0] - closest.xyz[0],
		viewpoint.xyz[1] - closest.xyz[1],
		viewpoint.xyz[2] - closest.xyz[2]
	};

	return sqrt(diff.xyz[0] * diff.xyz[0] + diff.xyz[1] * diff.xyz[1] + diff.xyz[2] * diff.xyz[2]);
}

__device__ float computeSizeGPU(Box& box, Point viewpoint, Point zdir)
{
	if (inboxCUDA(box, viewpoint))
		return FLT_MAX;

	float min_dist = pointboxdistCUDA(box, viewpoint);

	return box.minn.xyz[3] / min_dist;
}

__global__ void changeNodesOnce(
	Node* nodes,
	int N,
	int* indices,
	Box* boxes,
	Point* viewpoint,
	Point zdir,
	float target_size,
	int* split,
	int* node_counts,
	int* node_ids,
	char* needs_children
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 1; // repeat yourself
	char need_child = 0;
	if (size >= target_size)
	{
		if (node.depth > 0 && split[node_id] == 0) // split
		{
			if (node.start_children == -1)
			{
				node_ids[idx] = node_id;
				need_child = 1;
			}
			else
			{
				count += node.count_children;
				split[node_id] = 1;
			}
		}
	}
	else
	{
		int parent_node_id = node.parent;
		if (parent_node_id != -1)
		{
			// Node parent_node = nodes[parent_node_id];
			float parent_size = computeSizeGPU(boxes[parent_node_id], *viewpoint, zdir);
			if (parent_size < target_size) // collapse
			{
				split[parent_node_id] = 0;
				count = 0; // forget yourself
			}
		}
	}
	needs_children[idx] = need_child;
	node_counts[idx] = count;
}

__global__ void putNodes(
	Node* nodes,
	int N,
	int* indices,
	int* node_counts,
	int* node_offsets,
	int* next_nodes)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = node_counts[idx];
	if (count == 0)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];

	next_nodes[offset] = node_id;
	for (int i = 1; i < count; i++)
		next_nodes[offset + i] = node.start_children + i - 1;
}

__global__ void countRenderIndicesIndexed(Node* nodes, int* split, int N, int* node_indices, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = node.count_leafs;
	if (node.depth > 0 && split[node_idx] == 0)
		count += node.count_merged;

	render_counts[idx] = count;
}

__global__ void putRenderIndicesIndexed(Node* nodes, int N, int* node_indices, int* render_counts, int* render_offsets, int* render_indices, int* parent_indices, int* nodes_of_render_indices, Box* boxes, float3* debug)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = render_counts[idx];
	int offset = idx == 0 ? 0 : render_offsets[idx - 1];

	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		parent_indices[offset + i] = parentgaussian;
		nodes_of_render_indices[offset + i] = node_idx;
	}

	if (debug != nullptr)
	{
		// Box box = boxes[node_idx];
		for (int i = 0; i < count; i++)
		{
			float red = min(1.0f, node.depth / 10.0f);
			debug[offset + i] = { red, 1.0f - red, 0 };
			if (node.depth == 0)
				debug[offset + i] = { 0, 0, 1.0f };
		}
	}
}

void Switching::changeToSizeStep(
	float target_size,
	int N,
	int* node_indices,
	int* new_node_indices,
	int* nodes,
	float* boxes,
	float* viewpoint,
	float x, float y, float z,
	int* split,
	int* render_indices,
	int* parent_indices,
	int* nodes_of_render_indices,
	int* nodes_to_expand,
	float* debug,
	char*& scratchspace,
	size_t& scratchspacesize,
	int* NsrcI,
	int* NdstI,
	char* NdstC,
	int* numI,
	int maxN,
	int& add_success,
	int* new_N,
	int* new_R,
	int* need_expansion,
	void* maintenanceStream)
{
	hipStream_t stream = (hipStream_t)maintenanceStream;

	int num_node_blocks = (N + 255) / 256;

	Point zdir = { x, y, z };

	int* num_to_expand = numI;
	int* node_counts = NsrcI, * node_offsets = NdstI, * node_ids = NdstI;
	char* need_children = NdstC;
	if (scratchspacesize == 0)
	{
		size_t testsize;

		hipcub::DeviceScan::InclusiveSum(nullptr, testsize, node_counts, node_offsets, maxN, stream);
		scratchspacesize = testsize;
		hipcub::DeviceSelect::Flagged(nullptr, testsize, node_ids, need_children, nodes_to_expand, num_to_expand, maxN, stream);
		scratchspacesize = std::max(testsize, scratchspacesize);

		if (scratchspace)
			hipFree(scratchspace);
		scratchspacesize = testsize;
		hipMalloc(&scratchspace, scratchspacesize);
	}

	changeNodesOnce << <num_node_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		N, 
		node_indices, 
		(Box*)boxes, 
		(Point*)viewpoint, 
		zdir, 
		target_size, 
		split, 
		node_counts, 
		node_ids, 
		need_children
		);

	hipcub::DeviceSelect::Flagged(scratchspace, scratchspacesize, node_ids, need_children, nodes_to_expand, num_to_expand, N, stream);
	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, node_counts, node_offsets, N, stream);

	hipMemcpyAsync(need_expansion, num_to_expand, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(new_N, node_offsets + N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	if (*new_N > maxN)
	{
		add_success = 0;
		return;
	}

	putNodes << <num_node_blocks, 256, 0, stream>> > (
		(Node*)nodes,
		N, 
		node_indices, 
		node_counts, 
		node_offsets, 
		new_node_indices
		);

	int num_render_blocks = (*new_N + 255) / 256;
	int* render_counts = NsrcI, * render_offsets = NdstI;

	countRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		split, 
		*new_N, 
		new_node_indices, 
		render_counts
		);

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, render_counts, render_offsets, *new_N, stream);

	putRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		*new_N, 
		new_node_indices, 
		render_counts, 
		render_offsets, 
		render_indices, 
		parent_indices, 
		nodes_of_render_indices, 
		(Box*)boxes,
		(float3*)debug
		);

	hipMemcpyAsync(new_R, render_offsets + *new_N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);

	add_success = 1;
}

__global__ void markNodesForSize(Node* nodes, Box* boxes, int N, Point* viewpoint, Point zdir, float target_size, int* render_counts, int* node_markers)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = idx;
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 0;
	if (size >= target_size)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		float parent_size = computeSizeGPU(boxes[node.parent], *viewpoint, zdir);
		if (parent_size >= target_size)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}

	if (count != 0 && node_markers != nullptr)
		node_markers[node_id] = 1;

	if (render_counts != nullptr)
		render_counts[node_id] = count;
}

__global__ void computeTsIndexed(
	Node* nodes,
	Box* boxes,
	int N,
	int* indices,
	Point viewpoint,
	Point zdir,
	float target_size,
	float* ts,
	int* kids
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];

	float t;
	if (node.parent == -1)
		t = 1.0f;
	else
	{
		float parentsize = computeSizeGPU(boxes[node.parent], viewpoint, zdir);

		if (parentsize > 2.0f * target_size)
			t = 1.0f;
		else
		{
			float size = computeSizeGPU(boxes[node_id], viewpoint, zdir);
			float start = max(0.5f * parentsize, size);
			float diff = parentsize - start;

			if (diff <= 0)
				t = 1.0f;
			else
			{
				float tdiff = max(0.0f, target_size - start);
				t = max(1.0f - (tdiff / diff), 0.0f);
			}
		}
	}

	ts[idx] = t;
	kids[idx] = (node.parent == -1) ? 1 : nodes[node.parent].count_children;
}

void Switching::getTsIndexed(
	int N,
	int* indices,
	float target_size,
	int* nodes,
	float* boxes,
	float vx, float vy, float vz,
	float x, float y, float z,
	float* ts, // weights
	int* kids,
	void* stream
)
{
	Point zdir = { x, y, z };
	Point cam = { vx, vy, vz };
	int num_blocks = (N + 255) / 256;
	computeTsIndexed<<<num_blocks, 256, 0, (hipStream_t)stream >>>((Node*)nodes, (Box*)boxes, N, indices, cam, zdir, target_size, ts, kids);
}
__forceinline__ __device__ float3 tPoint4x3(const float3& p, const float* matrix)
{
	float3 transformed = {
		matrix[0] * p.x + matrix[4] * p.y + matrix[8] * p.z + matrix[12],
		matrix[1] * p.x + matrix[5] * p.y + matrix[9] * p.z + matrix[13],
		matrix[2] * p.x + matrix[6] * p.y + matrix[10] * p.z + matrix[14],
	};
	return transformed;
}

__forceinline__ __device__ float4 tPoint4x4(const float3& p, const float* matrix)
{
	float4 transformed = {
		matrix[0] * p.x + matrix[4] * p.y + matrix[8] * p.z + matrix[12],
		matrix[1] * p.x + matrix[5] * p.y + matrix[9] * p.z + matrix[13],
		matrix[2] * p.x + matrix[6] * p.y + matrix[10] * p.z + matrix[14],
		matrix[3] * p.x + matrix[7] * p.y + matrix[11] * p.z + matrix[15]
	};
	return transformed;
}

__forceinline__ __device__ bool frustum_culling(
	float3 p_orig,
	const float* viewmatrix,
	const float* projmatrix)
{
	float4 p_hom = tPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };
	float3 p_view = tPoint4x3(p_orig, viewmatrix);

	if (p_view.z <= 0.2f)// || ((p_proj.x < -1.3 || p_proj.x > 1.3 || p_proj.y < -1.3 || p_proj.y > 1.3)))
	{
		return false;
	}
	return true;
}

__global__ void expandToSize_markNodesForSize(int N, Node* nodes, Box* boxes, float* means3D, float threshold, Point* viewpoint, Point zdir,  
	int frame_index, int window_size, float* world_view_transform, float* projection_matrix, int* last_frame, int* render_counts) 
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];
	int index = node.start;
	float3 p_orig = { means3D[3 * index], means3D[3 * index + 1], means3D[3 * index + 2] };
	bool ret = frustum_culling(p_orig, world_view_transform, projection_matrix);
	// 视锥检查	
	if (last_frame[idx] >= frame_index - window_size || ((node.count_leafs + node.count_merged) != 1) || !ret){
		render_counts[idx] = 0;
		if (ret)
			last_frame[idx] = frame_index; // 更新 
		return ;
	} 
	Box box = boxes[idx];
	float size = computeSizeGPU(box, *viewpoint, zdir);
	int count = 0;
	if (size >= threshold) 
		count = node.count_leafs;
 	else { 
		float parent_size = computeSizeGPU(boxes[node.parent], *viewpoint, zdir);
		if (parent_size >= threshold) 
		{
			count = node.count_leafs;
			if (node.depth != 0) 
				count += node.count_merged;
		} 
	} 
	if(count > 0){ 
		render_counts[idx] = count; 
		last_frame[idx] = frame_index; 
	} 
} 

__global__ void expandToSize_putRenderIndices(int N, Node* nodes, int* child_indices, int* parent_indices, 
	int* child_box_indices, int* parent_box_indices, bool* leafs_tag, int* num_siblings, int* render_counts, int* render_offsets)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N || idx < 0 || render_counts[idx] != 1) 
		return;
	Node node = nodes[idx];
	int offset = (idx == 0) ? 0 : render_offsets[idx - 1];

	child_indices[offset] = node.start; // child's gs point index 
	parent_indices[offset] = nodes[node.parent].start; // parent's gs point index 
	child_box_indices[offset] = idx; // child's box index 
	parent_box_indices[offset] = node.parent; // parent's box index 
	leafs_tag[offset] = (node.count_leafs == 1);
	num_siblings[offset] = nodes[node.parent].count_children; 
}
int Switching::expandToSize(
	int N, 
	int* nodes, 
	float* boxes, 
	float* means3D, 
	float threshold, 
	float* viewpoint, 
	float x, float y, float z, 
	int frame_index, int window_size, 
	float* world_view_transform, 
	float* projection_matrix, 
	// list for clients 
	int* last_frame, 
	int* child_indices, 
	int* parent_indices, 
	int* child_box_indices, 
	int* parent_box_indices, 
	bool* leafs_tag,
	int* num_siblings) 
{
	// printf("start expand ---------------------------------------------------------- \n");
	// fflush(stdout);
	size_t temp_storage_bytes = 0;
	void* d_temp_storage = nullptr;
	int* render_counts = nullptr;
	int* render_offsets = nullptr;
	hipError_t err_1 = hipMalloc((void**)&render_counts, N * sizeof(int));
	hipError_t err_2 = hipMalloc((void**)&render_offsets, N * sizeof(int));
	if (err_1 != hipSuccess || err_2 != hipSuccess)
		printf("hipMalloc failed: %s, %s\n", hipGetErrorString(err_1), hipGetErrorString(err_2));
	Point zdir = { x, y, z };
	int num_blocks = (N + 255) / 256;
	expandToSize_markNodesForSize << <num_blocks, 256 >> > (N, (Node*)nodes, (Box*)boxes, means3D, threshold, (Point*)viewpoint, zdir, 
		frame_index, window_size, world_view_transform, projection_matrix, last_frame, render_counts);
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts, render_offsets, N);
	hipError_t err = hipMalloc(&d_temp_storage, temp_storage_bytes);
	if (err != hipSuccess)
		std::cout<<"hipMalloc failed: "<<hipGetErrorString(err)<<std::endl;
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, render_counts, render_offsets, N);
	expandToSize_putRenderIndices << <num_blocks, 256 >> > (N, (Node*)nodes, child_indices, parent_indices, 
		child_box_indices, parent_box_indices, leafs_tag, num_siblings, render_counts, render_offsets);

	int point_count = 0;
	hipMemcpy(&point_count, &(render_offsets[N-1]), sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_temp_storage);
	hipFree(render_counts);
	hipFree(render_offsets);
	return point_count;
}

void Switching::markVisibleForAllViewpoints(
	float target_size,
	int* nodes,
	int num_nodes,
	float* boxes,
	float* viewpoints,
	int num_viewpoints,
	int* seen,
	float zx,
	float zy,
	float zz
)
{
	thrust::device_vector<int> seen_cuda(num_nodes);
	thrust::device_vector<Point> viewpoint_cuda(1);
	thrust::device_vector<Node> nodes_cuda(num_nodes);
	thrust::device_vector<Box> boxes_cuda(num_nodes);

	hipMemcpy(nodes_cuda.data().get(), nodes, sizeof(Node) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(boxes_cuda.data().get(), boxes, sizeof(Box) * num_nodes, hipMemcpyHostToDevice);

	Point zdir = { zx, zy, zz };

	Point* points = (Point*)viewpoints;
	int num_blocks = (num_nodes + 255) / 256;
	for (int i = 0; i < num_viewpoints; i++)
	{
		Point viewpoint = points[i];
		hipMemcpy(viewpoint_cuda.data().get(), &viewpoint, sizeof(Point), hipMemcpyHostToDevice);

		markNodesForSize << <num_blocks, 256 >> > (
			nodes_cuda.data().get(),
			boxes_cuda.data().get(),
			num_nodes,
			viewpoint_cuda.data().get(),
			zdir,
			target_size,
			nullptr,
			seen_cuda.data().get());
	}
	hipMemcpy(seen, seen_cuda.data().get(), sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

	if (hipDeviceSynchronize())
		std::cout << "Errors: " << hipDeviceSynchronize() << std::endl;
}