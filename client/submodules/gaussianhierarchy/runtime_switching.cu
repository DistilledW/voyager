#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "hip/hip_runtime.h"
#include ""
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvToolsExt.h>
#include <thrust/host_vector.h>
#include <tuple>
#include "types.h"
#include "runtime_switching.h"

__global__ void markTargetNodes(Node* nodes, int N, int target, int* node_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = 0;
	Node node = nodes[idx];
	if (node.depth > target)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		Node parentnode = nodes[node.parent];
		if (parentnode.depth > target)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}
	node_counts[idx] = count;
}

__global__ void putRenderIndices(Node* nodes, int N, int* node_counts, int* node_offsets, int* render_indices, int* parent_indices = nullptr, int* nodes_for_render_indices = nullptr)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];
	int count = node_counts[idx];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];
	int start = node.start;
	
	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		if (parent_indices)
			parent_indices[offset + i] = parentgaussian; 
		if (nodes_for_render_indices)
			nodes_for_render_indices[offset + i] = idx;
	}
}

int Switching::expandToTarget(
	int N,
	int target,
	int* nodes,
	int* render_indices
)
{
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);

	int num_blocks = (N + 255) / 256;
	markTargetNodes << <num_blocks, 256 >> > ((Node*)nodes, N, target, render_counts.data().get());

	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);

	putRenderIndices << <num_blocks, 256 >> > ((Node*)nodes, N, render_counts.data().get(), render_offsets.data().get(), render_indices);

	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

__device__ bool inboxCUDA(Box& box, Point viewpoint)
{
	bool inside = true;
	for (int i = 0; i < 3; i++)
	{
		inside &= viewpoint.xyz[i] >= box.minn.xyz[i] && viewpoint.xyz[i] <= box.maxx.xyz[i];
	}
	return inside;
}

__device__ float pointboxdistCUDA(Box& box, Point viewpoint)
{
	Point closest = {
		max(box.minn.xyz[0], min(box.maxx.xyz[0], viewpoint.xyz[0])),
		max(box.minn.xyz[1], min(box.maxx.xyz[1], viewpoint.xyz[1])),
		max(box.minn.xyz[2], min(box.maxx.xyz[2], viewpoint.xyz[2]))
	};

	Point diff = {
		viewpoint.xyz[0] - closest.xyz[0],
		viewpoint.xyz[1] - closest.xyz[1],
		viewpoint.xyz[2] - closest.xyz[2]
	};

	return sqrt(diff.xyz[0] * diff.xyz[0] + diff.xyz[1] * diff.xyz[1] + diff.xyz[2] * diff.xyz[2]);
}

__device__ float computeSizeGPU(Box& box, Point viewpoint, Point zdir)
{
	if (inboxCUDA(box, viewpoint))
		return FLT_MAX;

	float min_dist = pointboxdistCUDA(box, viewpoint);

	return box.minn.xyz[3] / min_dist;
}

__global__ void changeNodesOnce(
	Node* nodes,
	int N,
	int* indices,
	Box* boxes,
	Point* viewpoint,
	Point zdir,
	float target_size,
	int* split,
	int* node_counts,
	int* node_ids,
	char* needs_children
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 1; // repeat yourself
	char need_child = 0;
	if (size >= target_size)
	{
		if (node.depth > 0 && split[node_id] == 0) // split
		{
			if (node.start_children == -1)
			{
				node_ids[idx] = node_id;
				need_child = 1;
			}
			else
			{
				count += node.count_children;
				split[node_id] = 1;
			}
		}
	}
	else
	{
		int parent_node_id = node.parent;
		if (parent_node_id != -1)
		{
			Node parent_node = nodes[parent_node_id];
			float parent_size = computeSizeGPU(boxes[parent_node_id], *viewpoint, zdir);
			if (parent_size < target_size) // collapse
			{
				split[parent_node_id] = 0;
				count = 0; // forget yourself
			}
		}
	}
	needs_children[idx] = need_child;
	node_counts[idx] = count;
}

__global__ void putNodes(
	Node* nodes,
	int N,
	int* indices,
	int* node_counts,
	int* node_offsets,
	int* next_nodes)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = node_counts[idx];
	if (count == 0)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];

	next_nodes[offset] = node_id;
	for (int i = 1; i < count; i++)
		next_nodes[offset + i] = node.start_children + i - 1;
}

__global__ void countRenderIndicesIndexed(Node* nodes, int* split, int N, int* node_indices, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = node.count_leafs;
	if (node.depth > 0 && split[node_idx] == 0)
		count += node.count_merged;

	render_counts[idx] = count;
}

__global__ void putRenderIndicesIndexed(Node* nodes, int N, int* node_indices, int* render_counts, int* render_offsets, int* render_indices, int* parent_indices, int* nodes_of_render_indices, Box* boxes, float3* debug)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = render_counts[idx];
	int offset = idx == 0 ? 0 : render_offsets[idx - 1];
	// int start = node.start;

	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		parent_indices[offset + i] = parentgaussian;
		nodes_of_render_indices[offset + i] = node_idx;
	}

	if (debug != nullptr)
	{
		Box box = boxes[node_idx];
		for (int i = 0; i < count; i++)
		{
			float red = min(1.0f, node.depth / 10.0f);
			debug[offset + i] = { red, 1.0f - red, 0 };
			if (node.depth == 0)
				debug[offset + i] = { 0, 0, 1.0f };
		}
	}
}

void Switching::changeToSizeStep(
	float target_size,
	int N,
	int* node_indices,
	int* new_node_indices,
	int* nodes,
	float* boxes,
	float* viewpoint,
	float x, float y, float z,
	int* split,
	int* render_indices,
	int* parent_indices,
	int* nodes_of_render_indices,
	int* nodes_to_expand,
	float* debug,
	char*& scratchspace,
	size_t& scratchspacesize,
	int* NsrcI,
	int* NdstI,
	char* NdstC,
	int* numI,
	int maxN,
	int& add_success,
	int* new_N,
	int* new_R,
	int* need_expansion,
	void* maintenanceStream)
{
	hipStream_t stream = (hipStream_t)maintenanceStream;

	int num_node_blocks = (N + 255) / 256;

	Point zdir = { x, y, z };

	int* num_to_expand = numI;
	int* node_counts = NsrcI, * node_offsets = NdstI, * node_ids = NdstI;
	char* need_children = NdstC;
	if (scratchspacesize == 0)
	{
		size_t testsize;

		hipcub::DeviceScan::InclusiveSum(nullptr, testsize, node_counts, node_offsets, maxN, stream);
		scratchspacesize = testsize;
		hipcub::DeviceSelect::Flagged(nullptr, testsize, node_ids, need_children, nodes_to_expand, num_to_expand, maxN, stream);
		scratchspacesize = std::max(testsize, scratchspacesize);

		if (scratchspace)
			hipFree(scratchspace);
		scratchspacesize = testsize;
		hipMalloc(&scratchspace, scratchspacesize);
	}

	changeNodesOnce << <num_node_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		N, 
		node_indices, 
		(Box*)boxes, 
		(Point*)viewpoint, 
		zdir, 
		target_size, 
		split, 
		node_counts, 
		node_ids, 
		need_children
		);

	hipcub::DeviceSelect::Flagged(scratchspace, scratchspacesize, node_ids, need_children, nodes_to_expand, num_to_expand, N, stream);
	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, node_counts, node_offsets, N, stream);

	hipMemcpyAsync(need_expansion, num_to_expand, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(new_N, node_offsets + N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	if (*new_N > maxN)
	{
		add_success = 0;
		return;
	}

	putNodes << <num_node_blocks, 256, 0, stream>> > (
		(Node*)nodes,
		N, 
		node_indices, 
		node_counts, 
		node_offsets, 
		new_node_indices
		);

	int num_render_blocks = (*new_N + 255) / 256;
	int* render_counts = NsrcI, * render_offsets = NdstI;

	countRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		split, 
		*new_N, 
		new_node_indices, 
		render_counts
		);

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, render_counts, render_offsets, *new_N, stream);

	putRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		*new_N, 
		new_node_indices, 
		render_counts, 
		render_offsets, 
		render_indices, 
		parent_indices, 
		nodes_of_render_indices, 
		(Box*)boxes,
		(float3*)debug
		);

	hipMemcpyAsync(new_R, render_offsets + *new_N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);

	add_success = 1;
}

__global__ void markNodesForSize(Node* nodes, Box* boxes, int N, Point* viewpoint, Point zdir, float target_size, int* render_counts, int* node_markers)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = idx;
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 0;
	if (size >= target_size)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		float parent_size = computeSizeGPU(boxes[node.parent], *viewpoint, zdir);
		if (parent_size >= target_size)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}

	if (count != 0 && node_markers != nullptr)
		node_markers[node_id] = 1;

	if (render_counts != nullptr)
		render_counts[node_id] = count;
}

__global__ void computeTsIndexed(
	Box* child_boxes,
	Box* parent_boxes,
	int N,
	int* indices,
	Point viewpoint,
	Point zdir,
	float target_size,
	float* ts //,
	// int* kids
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];

	float t;
	if (parent_boxes[node_id].minn.xyz[0] == parent_boxes[node_id].maxx.xyz[0] && parent_boxes[node_id].minn.xyz[0] == 0) 
		t = 1.0f; // parent boxes 不存在
	else
	{
		float parentsize = computeSizeGPU(parent_boxes[node_id], viewpoint, zdir);

		if (parentsize > 2.0f * target_size)
			t = 1.0f;
		else
		{
			float size = computeSizeGPU(child_boxes[node_id], viewpoint, zdir);
			float start = max(0.5f * parentsize, size);
			float diff = parentsize - start;

			if (diff <= 0)
				t = 1.0f;
			else
			{
				float tdiff = max(0.0f, target_size - start);
				t = max(1.0f - (tdiff / diff), 0.0f);
			}
		}
	}

	ts[idx] = t;
	// kids[idx] = (node.parent == -1) ? 1 : nodes[node.parent].count_children;
}

void Switching::getTsIndexed(
	int N,
	int* indices,
	float target_size,
	float* child_boxes,
	float* parent_boxes,
	float vx, float vy, float vz,
	float x, float y, float z,
	float* ts,
	// int* kids,
	void* stream
)
{
	Point zdir = { x, y, z };
	Point cam = { vx, vy, vz };
	int num_blocks = (N + 255) / 256;
	computeTsIndexed<<<num_blocks, 256, 0, (hipStream_t)stream >>>(
		(Box*)child_boxes, 
		(Box*)parent_boxes, 
		N, 
		indices, 
		cam,
		zdir, 
		target_size, 
		ts); // , kids
		
}
__global__ void expand_markNodesForSize(int N, Box* c_boxes, Box*p_boxes, int threshold, Point* viewpoint, Point zdir, bool if_culling, GSPlane* frustum_plans, bool* leafs_tag, int* last_frame, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N) 
		return;
	last_frame[idx]++;
	int curr_idx = idx;
	// 视锥检查，过滤掉所有在视锥外的点
	Box box = c_boxes[curr_idx];
	if (if_culling){
		for(int i = 0; i < 6; ++i){
			GSPlane& plane = frustum_plans[i];
			bool allOutSide = true;
			for (int j = 0; j < 8; ++j){ 
				float x = (j & 1 == 0) ? box.minn.xyz[0] : box.maxx.xyz[0] ;
				float y = (j & 2 == 0) ? box.minn.xyz[1] : box.maxx.xyz[1] ;
				float z = (j & 4 == 0) ? box.minn.xyz[2] : box.maxx.xyz[2] ;
				float dist = x * plane.x + y * plane.y + z * plane.z + plane.d;
				if (dist < 0){ 
					allOutSide = false;
					break;
				}
			}
			if(allOutSide){
				render_counts[idx] = 0;
				return ;
			}
		} 
	}
	float size = computeSizeGPU(c_boxes[curr_idx], *viewpoint, zdir);	// child size
	if (size >= threshold){
		// if (leafs_tag[curr_idx]){
			render_counts[curr_idx] = 1;
			last_frame[curr_idx] = 0;
		// }
	}else {
		float p_size = computeSizeGPU(p_boxes[curr_idx], *viewpoint, zdir);	// parent size
		if(p_size >= threshold){
			render_counts[curr_idx] = 1;
			last_frame[curr_idx] = 0;
		}
	} 
}
__global__ void expand_putRenderIndices(int N, Box* c_boxes, Box* p_boxes, float threshold, Point* viewpoint, Point zdir, int*render_indices, float* interpolation_weights, int*render_counts, int*render_offsets)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N || render_counts[idx] <= 0) 
		return;
	int curr_idx = idx;
	int offset = render_offsets[curr_idx];
	
	float c_size = computeSizeGPU(c_boxes[curr_idx], *viewpoint, zdir);	// child size 
	float p_size = computeSizeGPU(p_boxes[curr_idx], *viewpoint, zdir);	// parent size 
	float t = 1.0f, start, diff;

	if (p_size >= 2.0f * threshold){
		t = 1.0f;
	}else{
		start = max(0.5f * p_size, c_size);
		diff = p_size - start;
		if (diff <= 1.0f)
			t = 1.0f;
		else{
			float tdiff = max(0.0f, threshold - start);
			t = max(1.0f - (tdiff / diff), 0.0f);
		}
	}
	render_indices[offset] = curr_idx;
	interpolation_weights[offset] = t;
}

int Switching::forceSearch(
	int N, 						// point number
	float* c_boxes, 			// children's boxes
	float* p_boxes, 			// parents' boxes
	float threshold, 			// target size
	float* viewpoint,
	float x, float y, float z,
	float* frustum_plans, 		// 视锥平面 
	bool* leafs_tag, 
	bool if_culling,
	// Output 
	int* last_frame,			// 上一次用到该 gs point有多久。
	int* render_indices, 
	float* interpolation_weights) 
{
	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);
	Point zdir = { x, y, z };
	int num_blocks = (N + 255) / 256;
	expand_markNodesForSize << <num_blocks, 256 >> > (N, (Box*)c_boxes, (Box*)p_boxes, threshold, (Point*)viewpoint, zdir, if_culling, (GSPlane*)frustum_plans, 
		(bool*)leafs_tag, last_frame, render_counts.data().get());
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	expand_putRenderIndices << <num_blocks, 256 >> > (N, (Box*)c_boxes, (Box*)p_boxes, threshold, (Point*)viewpoint, zdir, render_indices, 
		interpolation_weights, render_counts.data().get(), render_offsets.data().get());
	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

void Switching::markVisibleForAllViewpoints(
	float target_size,
	int* nodes,
	int num_nodes,
	float* boxes,
	float* viewpoints,
	int num_viewpoints,
	int* seen,
	float zx,
	float zy,
	float zz
)
{
	thrust::device_vector<int> seen_cuda(num_nodes);
	thrust::device_vector<Point> viewpoint_cuda(1);
	thrust::device_vector<Node> nodes_cuda(num_nodes);
	thrust::device_vector<Box> boxes_cuda(num_nodes);

	hipMemcpy(nodes_cuda.data().get(), nodes, sizeof(Node) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(boxes_cuda.data().get(), boxes, sizeof(Box) * num_nodes, hipMemcpyHostToDevice);

	Point zdir = { zx, zy, zz };

	Point* points = (Point*)viewpoints;
	int num_blocks = (num_nodes + 255) / 256;
	for (int i = 0; i < num_viewpoints; i++)
	{
		Point viewpoint = points[i];
		hipMemcpy(viewpoint_cuda.data().get(), &viewpoint, sizeof(Point), hipMemcpyHostToDevice);

		markNodesForSize << <num_blocks, 256 >> > (
			nodes_cuda.data().get(),
			boxes_cuda.data().get(),
			num_nodes,
			viewpoint_cuda.data().get(),
			zdir,
			target_size,
			nullptr,
			seen_cuda.data().get());
	}
	hipMemcpy(seen, seen_cuda.data().get(), sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

	if (hipDeviceSynchronize())
		std::cout << "Errors: " << hipDeviceSynchronize() << std::endl;
}