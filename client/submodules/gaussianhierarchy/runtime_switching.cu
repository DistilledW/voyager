#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2024, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "hip/hip_runtime.h"
#include ""
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <iostream>
#include <algorithm>
#include <numeric>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvToolsExt.h>
#include <thrust/host_vector.h>
#include <tuple>
#include "types.h"
#include "runtime_switching.h"

__global__ void markTargetNodes(Node* nodes, int N, int target, int* node_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = 0;
	Node node = nodes[idx];
	if (node.depth > target)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		Node parentnode = nodes[node.parent];
		if (parentnode.depth > target)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}
	node_counts[idx] = count;
}

__global__ void putRenderIndices(Node* nodes, int N, int* node_counts, int* node_offsets, int* render_indices, int* parent_indices = nullptr, int* nodes_for_render_indices = nullptr)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	Node node = nodes[idx];
	int count = node_counts[idx];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];
	int start = node.start;
	
	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		if (parent_indices)
			parent_indices[offset + i] = parentgaussian; 
		if (nodes_for_render_indices)
			nodes_for_render_indices[offset + i] = idx;
	}
}

int Switching::expandToTarget(
	int N,
	int target,
	int* nodes,
	int* render_indices
)
{
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);

	int num_blocks = (N + 255) / 256;
	markTargetNodes << <num_blocks, 256 >> > ((Node*)nodes, N, target, render_counts.data().get());

	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);

	putRenderIndices << <num_blocks, 256 >> > ((Node*)nodes, N, render_counts.data().get(), render_offsets.data().get(), render_indices);

	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

__device__ bool inboxCUDA(Box& box, Point viewpoint)
{
	bool inside = true;
	for (int i = 0; i < 3; i++)
	{
		inside &= viewpoint.xyz[i] >= box.minn.xyz[i] && viewpoint.xyz[i] <= box.maxx.xyz[i];
	}
	return inside;
}

__device__ float pointboxdistCUDA(Box& box, Point viewpoint)
{
	Point closest = {
		max(box.minn.xyz[0], min(box.maxx.xyz[0], viewpoint.xyz[0])),
		max(box.minn.xyz[1], min(box.maxx.xyz[1], viewpoint.xyz[1])),
		max(box.minn.xyz[2], min(box.maxx.xyz[2], viewpoint.xyz[2]))
	};

	Point diff = {
		viewpoint.xyz[0] - closest.xyz[0],
		viewpoint.xyz[1] - closest.xyz[1],
		viewpoint.xyz[2] - closest.xyz[2]
	};

	return sqrt(diff.xyz[0] * diff.xyz[0] + diff.xyz[1] * diff.xyz[1] + diff.xyz[2] * diff.xyz[2]);
}

__device__ float computeSizeGPU(Box& box, Point viewpoint, Point zdir)
{
	if (inboxCUDA(box, viewpoint))
		return FLT_MAX;

	float min_dist = pointboxdistCUDA(box, viewpoint);

	return box.minn.xyz[3] / min_dist;
}

__global__ void changeNodesOnce(
	Node* nodes,
	int N,
	int* indices,
	Box* boxes,
	Point* viewpoint,
	Point zdir,
	float target_size,
	int* split,
	int* node_counts,
	int* node_ids,
	char* needs_children
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 1; // repeat yourself
	char need_child = 0;
	if (size >= target_size)
	{
		if (node.depth > 0 && split[node_id] == 0) // split
		{
			if (node.start_children == -1)
			{
				node_ids[idx] = node_id;
				need_child = 1;
			}
			else
			{
				count += node.count_children;
				split[node_id] = 1;
			}
		}
	}
	else
	{
		int parent_node_id = node.parent;
		if (parent_node_id != -1)
		{
			Node parent_node = nodes[parent_node_id];
			float parent_size = computeSizeGPU(boxes[parent_node_id], *viewpoint, zdir);
			if (parent_size < target_size) // collapse
			{
				split[parent_node_id] = 0;
				count = 0; // forget yourself
			}
		}
	}
	needs_children[idx] = need_child;
	node_counts[idx] = count;
}

__global__ void putNodes(
	Node* nodes,
	int N,
	int* indices,
	int* node_counts,
	int* node_offsets,
	int* next_nodes)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int count = node_counts[idx];
	if (count == 0)
		return;

	int node_id = indices[idx];
	Node node = nodes[node_id];
	int offset = idx == 0 ? 0 : node_offsets[idx - 1];

	next_nodes[offset] = node_id;
	for (int i = 1; i < count; i++)
		next_nodes[offset + i] = node.start_children + i - 1;
}

__global__ void countRenderIndicesIndexed(Node* nodes, int* split, int N, int* node_indices, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = node.count_leafs;
	if (node.depth > 0 && split[node_idx] == 0)
		count += node.count_merged;

	render_counts[idx] = count;
}

__global__ void putRenderIndicesIndexed(Node* nodes, int N, int* node_indices, int* render_counts, int* render_offsets, int* render_indices, int* parent_indices, int* nodes_of_render_indices, Box* boxes, float3* debug)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_idx = node_indices[idx];

	Node node = nodes[node_idx];
	int count = render_counts[idx];
	int offset = idx == 0 ? 0 : render_offsets[idx - 1];
	// int start = node.start;

	int parentgaussian = -1;
	if (node.parent != -1)
	{
		parentgaussian = nodes[node.parent].start;
	}

	for (int i = 0; i < count; i++)
	{
		render_indices[offset + i] = node.start + i;
		parent_indices[offset + i] = parentgaussian;
		nodes_of_render_indices[offset + i] = node_idx;
	}

	if (debug != nullptr)
	{
		Box box = boxes[node_idx];
		for (int i = 0; i < count; i++)
		{
			float red = min(1.0f, node.depth / 10.0f);
			debug[offset + i] = { red, 1.0f - red, 0 };
			if (node.depth == 0)
				debug[offset + i] = { 0, 0, 1.0f };
		}
	}
}

void Switching::changeToSizeStep(
	float target_size,
	int N,
	int* node_indices,
	int* new_node_indices,
	int* nodes,
	float* boxes,
	float* viewpoint,
	float x, float y, float z,
	int* split,
	int* render_indices,
	int* parent_indices,
	int* nodes_of_render_indices,
	int* nodes_to_expand,
	float* debug,
	char*& scratchspace,
	size_t& scratchspacesize,
	int* NsrcI,
	int* NdstI,
	char* NdstC,
	int* numI,
	int maxN,
	int& add_success,
	int* new_N,
	int* new_R,
	int* need_expansion,
	void* maintenanceStream)
{
	hipStream_t stream = (hipStream_t)maintenanceStream;

	int num_node_blocks = (N + 255) / 256;

	Point zdir = { x, y, z };

	int* num_to_expand = numI;
	int* node_counts = NsrcI, * node_offsets = NdstI, * node_ids = NdstI;
	char* need_children = NdstC;
	if (scratchspacesize == 0)
	{
		size_t testsize;

		hipcub::DeviceScan::InclusiveSum(nullptr, testsize, node_counts, node_offsets, maxN, stream);
		scratchspacesize = testsize;
		hipcub::DeviceSelect::Flagged(nullptr, testsize, node_ids, need_children, nodes_to_expand, num_to_expand, maxN, stream);
		scratchspacesize = std::max(testsize, scratchspacesize);

		if (scratchspace)
			hipFree(scratchspace);
		scratchspacesize = testsize;
		hipMalloc(&scratchspace, scratchspacesize);
	}

	changeNodesOnce << <num_node_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		N, 
		node_indices, 
		(Box*)boxes, 
		(Point*)viewpoint, 
		zdir, 
		target_size, 
		split, 
		node_counts, 
		node_ids, 
		need_children
		);

	hipcub::DeviceSelect::Flagged(scratchspace, scratchspacesize, node_ids, need_children, nodes_to_expand, num_to_expand, N, stream);
	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, node_counts, node_offsets, N, stream);

	hipMemcpyAsync(need_expansion, num_to_expand, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(new_N, node_offsets + N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);

	if (*new_N > maxN)
	{
		add_success = 0;
		return;
	}

	putNodes << <num_node_blocks, 256, 0, stream>> > (
		(Node*)nodes,
		N, 
		node_indices, 
		node_counts, 
		node_offsets, 
		new_node_indices
		);

	int num_render_blocks = (*new_N + 255) / 256;
	int* render_counts = NsrcI, * render_offsets = NdstI;

	countRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		split, 
		*new_N, 
		new_node_indices, 
		render_counts
		);

	hipcub::DeviceScan::InclusiveSum(scratchspace, scratchspacesize, render_counts, render_offsets, *new_N, stream);

	putRenderIndicesIndexed << <num_render_blocks, 256, 0, stream >> > (
		(Node*)nodes, 
		*new_N, 
		new_node_indices, 
		render_counts, 
		render_offsets, 
		render_indices, 
		parent_indices, 
		nodes_of_render_indices, 
		(Box*)boxes,
		(float3*)debug
		);

	hipMemcpyAsync(new_R, render_offsets + *new_N - 1, sizeof(int), hipMemcpyDeviceToHost, stream);

	add_success = 1;
}

__global__ void markNodesForSize(Node* nodes, Box* boxes, int N, Point* viewpoint, Point zdir, float target_size, int* render_counts, int* node_markers)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = idx;
	Node node = nodes[node_id];
	float size = computeSizeGPU(boxes[node_id], *viewpoint, zdir);

	int count = 0;
	if (size >= target_size)
		count = node.count_leafs;
	else if (node.parent != -1)
	{
		float parent_size = computeSizeGPU(boxes[node.parent], *viewpoint, zdir);
		if (parent_size >= target_size)
		{
			count = node.count_leafs;
			if (node.depth != 0)
				count += node.count_merged;
		}
	}

	if (count != 0 && node_markers != nullptr)
		node_markers[node_id] = 1;

	if (render_counts != nullptr)
		render_counts[node_id] = count;
}

__global__ void computeTsIndexed(
	Box* child_boxes,
	Box* parent_boxes,
	int N,
	int* indices,
	Point viewpoint,
	Point zdir,
	float target_size,
	float* ts //,
	// int* kids
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N)
		return;

	int node_id = indices[idx];

	float t;
	if (parent_boxes[node_id].minn.xyz[0] == parent_boxes[node_id].maxx.xyz[0] && parent_boxes[node_id].minn.xyz[0] == 0) 
		t = 1.0f; // parent boxes 不存在
	else
	{
		float parentsize = computeSizeGPU(parent_boxes[node_id], viewpoint, zdir);

		if (parentsize > 2.0f * target_size)
			t = 1.0f;
		else
		{
			float size = computeSizeGPU(child_boxes[node_id], viewpoint, zdir);
			float start = max(0.5f * parentsize, size);
			float diff = parentsize - start;

			if (diff <= 0)
				t = 1.0f;
			else
			{
				float tdiff = max(0.0f, target_size - start);
				t = max(1.0f - (tdiff / diff), 0.0f);
			}
		}
	}

	ts[idx] = t;
	// kids[idx] = (node.parent == -1) ? 1 : nodes[node.parent].count_children;
}

void Switching::getTsIndexed(
	int N,
	int* indices,
	float target_size,
	float* child_boxes,
	float* parent_boxes,
	float vx, float vy, float vz,
	float x, float y, float z,
	float* ts,
	// int* kids,
	void* stream
)
{
	Point zdir = { x, y, z };
	Point cam = { vx, vy, vz };
	int num_blocks = (N + 255) / 256;
	computeTsIndexed<<<num_blocks, 256, 0, (hipStream_t)stream >>>(
		(Box*)child_boxes, 
		(Box*)parent_boxes, 
		N, 
		indices, 
		cam,
		zdir, 
		target_size, 
		ts); // , kids
		
}
__forceinline__ __device__ float3 tPoint4x3(const float3& p, const float* matrix)
{
	float3 transformed = {
		matrix[0] * p.x + matrix[4] * p.y + matrix[8] * p.z + matrix[12],
		matrix[1] * p.x + matrix[5] * p.y + matrix[9] * p.z + matrix[13],
		matrix[2] * p.x + matrix[6] * p.y + matrix[10] * p.z + matrix[14],
	};
	return transformed;
}

__forceinline__ __device__ float4 tPoint4x4(const float3& p, const float* matrix)
{
	float4 transformed = {
		matrix[0] * p.x + matrix[4] * p.y + matrix[8] * p.z + matrix[12],
		matrix[1] * p.x + matrix[5] * p.y + matrix[9] * p.z + matrix[13],
		matrix[2] * p.x + matrix[6] * p.y + matrix[10] * p.z + matrix[14],
		matrix[3] * p.x + matrix[7] * p.y + matrix[11] * p.z + matrix[15]
	};
	return transformed;
}

__forceinline__ __device__ bool in_frustum(
	float3 p_orig,
	const float* viewmatrix,
	const float* projmatrix)
{
	// float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };

	// Bring points to screen space
	float4 p_hom = tPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };
	float3 p_view = tPoint4x3(p_orig, viewmatrix);

	if (p_view.z <= 0.2f)// || ((p_proj.x < -1.3 || p_proj.x > 1.3 || p_proj.y < -1.3 || p_proj.y > 1.3)))
	{
		return false;
	}
	return true;
}

__global__ void expand_markNodesForSize(int N, Box* c_boxes, Box*p_boxes, float* means3d, int threshold, Point* viewpoint, Point zdir, bool if_culling, float* view_transform, float*  projection_matrix, bool* leafs_tag, int* last_frame, int* render_counts)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N) 
		return;
	last_frame[idx]++;
	int curr_idx = idx;
	// 视锥检查，过滤掉所有在视锥外的点
	Box box = c_boxes[curr_idx];
	if (if_culling){
		float3 p_orig = { means3d[3 * idx], means3d[3 * idx + 1], means3d[3 * idx + 2] };
		if (!in_frustum(p_orig, view_transform, projection_matrix)){
			render_counts[idx] = 0;
			return ;
		}
	}
	float size = computeSizeGPU(c_boxes[curr_idx], *viewpoint, zdir);	// child size
	if (size >= threshold){
		// if (leafs_tag[curr_idx]){
			render_counts[curr_idx] = 1;
			last_frame[curr_idx] = 0;
		// }
	}else {
		float p_size = computeSizeGPU(p_boxes[curr_idx], *viewpoint, zdir);	// parent size
		if(p_size >= threshold){
			render_counts[curr_idx] = 1;
			last_frame[curr_idx] = 0;
		}
	} 
}
__global__ void expand_putRenderIndices(int N, Box* c_boxes, Box* p_boxes, float threshold, Point* viewpoint, Point zdir, int*render_indices, float* interpolation_weights, int*render_counts, int*render_offsets)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= N || render_counts[idx] <= 0) 
		return;
	int curr_idx = idx;
	int offset = render_offsets[curr_idx];
	
	float c_size = computeSizeGPU(c_boxes[curr_idx], *viewpoint, zdir);	// child size 
	float p_size = computeSizeGPU(p_boxes[curr_idx], *viewpoint, zdir);	// parent size 
	float t = 1.0f, start, diff;

	if (p_size >= 2.0f * threshold){
		t = 1.0f;
	}else{
		start = max(0.5f * p_size, c_size);
		diff = p_size - start;
		if (diff <= 1.0f)
			t = 1.0f;
		else{
			float tdiff = max(0.0f, threshold - start);
			t = max(1.0f - (tdiff / diff), 0.0f);
		}
	}
	render_indices[offset] = curr_idx;
	interpolation_weights[offset] = t;
}

int Switching::forceSearch(
	int N, 						// point number
	float* c_boxes, 			// children's boxes
	float* p_boxes, 			// parents' boxes
	float* means3d, 
	float threshold, 			// target size
	float* viewpoint,
	float x, float y, float z,
	bool if_culling,
	float* view_transform,
	float* projection_matrix,	
	bool* leafs_tag, 
	// Output 
	int* last_frame,			// 上一次用到该 gs point有多久。
	int* render_indices, 
	float* interpolation_weights) 
{
	size_t temp_storage_bytes;
	thrust::device_vector<char> temp_storage;
	thrust::device_vector<int> render_counts(N);
	thrust::device_vector<int> render_offsets(N);
	Point zdir = { x, y, z };
	int num_blocks = (N + 255) / 256;
	expand_markNodesForSize << <num_blocks, 256 >> > (N, (Box*)c_boxes, (Box*)p_boxes, means3d, threshold, (Point*)viewpoint, zdir, if_culling, \
		view_transform, projection_matrix, (bool*)leafs_tag, last_frame, render_counts.data().get());
	hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	temp_storage.resize(temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(temp_storage.data().get(), temp_storage_bytes, render_counts.data().get(), render_offsets.data().get(), N);
	expand_putRenderIndices << <num_blocks, 256 >> > (N, (Box*)c_boxes, (Box*)p_boxes, threshold, (Point*)viewpoint, zdir, render_indices, 
		interpolation_weights, render_counts.data().get(), render_offsets.data().get());
	int count = 0;
	hipMemcpy(&count, render_offsets.data().get() + N - 1, sizeof(int), hipMemcpyDeviceToHost);
	return count;
}

void Switching::markVisibleForAllViewpoints(
	float target_size,
	int* nodes,
	int num_nodes,
	float* boxes,
	float* viewpoints,
	int num_viewpoints,
	int* seen,
	float zx,
	float zy,
	float zz
)
{
	thrust::device_vector<int> seen_cuda(num_nodes);
	thrust::device_vector<Point> viewpoint_cuda(1);
	thrust::device_vector<Node> nodes_cuda(num_nodes);
	thrust::device_vector<Box> boxes_cuda(num_nodes);

	hipMemcpy(nodes_cuda.data().get(), nodes, sizeof(Node) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(boxes_cuda.data().get(), boxes, sizeof(Box) * num_nodes, hipMemcpyHostToDevice);

	Point zdir = { zx, zy, zz };

	Point* points = (Point*)viewpoints;
	int num_blocks = (num_nodes + 255) / 256;
	for (int i = 0; i < num_viewpoints; i++)
	{
		Point viewpoint = points[i];
		hipMemcpy(viewpoint_cuda.data().get(), &viewpoint, sizeof(Point), hipMemcpyHostToDevice);

		markNodesForSize << <num_blocks, 256 >> > (
			nodes_cuda.data().get(),
			boxes_cuda.data().get(),
			num_nodes,
			viewpoint_cuda.data().get(),
			zdir,
			target_size,
			nullptr,
			seen_cuda.data().get());
	}
	hipMemcpy(seen, seen_cuda.data().get(), sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

	if (hipDeviceSynchronize())
		std::cout << "Errors: " << hipDeviceSynchronize() << std::endl;
}